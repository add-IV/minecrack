#include "hip/hip_runtime.h"
#include "cub/iterator/cache_modified_input_iterator.cuh"
#include "utilities.hpp"
#include "utilities_cuda.cuh"
#include "JavaRandom.cuh"
#include "minecrack-48bit.hpp"

hipError_t hipOccupancyMaxPotentialBlockSize(int *minGridSize, int *blockSize, const void *func, size_t dynamicSMemSize,
                                             int blockSizeLimit) {
	return hipOccupancyMaxPotentialBlockSize<const void *>(minGridSize, blockSize, func, dynamicSMemSize,
			blockSizeLimit);
}

namespace GPU {

__device__ int64_t seed_offsets[MAX_SLIME_CHUNKS];
__constant__ uint8_t seed_offsets_len;

__global__ void test_seeds(uint64_t seeds_base, uint64_t seeds_max, uint64_t* passed_seeds) {
	static uint32_t passed_buffer_i = 0;
	hipcub::CacheModifiedInputIterator<hipcub::LOAD_LDG, int64_t> seed_offsets(GPU::seed_offsets);
	auto threads = gridDim.x * blockDim.x, threadidx = blockDim.x * blockIdx.x + threadIdx.x;
	uint32_t seeds_total = seeds_max - seeds_base, seeds_portion = seeds_total / threads;
	seeds_base += seeds_portion * threadidx;
	seeds_max = seeds_base + seeds_portion + (threadidx == threads - 1 ? seeds_total % threads : 0);

	/*
	 * To flatten the flow control and minimize thread divergence, we run seed_offsets_len tests on some current_seed:
	 * when a test fails, the current_seed is immediately replaced with another one, otherwise if all tests succeed the
	 * seed is not immediately put into the output buffer but it is saved in good_seed. The value of good_seed is
	 * flushed to the output after targets_len iterations of the test. In this way the test loops has less control flow
	 * branches, and all of them can generally be implemented with fast predicates. The test loop is long exactly
	 * targets_len so that only one good_seed can be generated, otherwise multiple seeds could pass the tests and
	 * good_seed could be overwritten before flushing the result to output.
	 */
	constexpr const uint64_t bad_seed = 1ULL << JavaRandom::generator_bits;
	uint64_t current_seed = bad_seed, good_seed = bad_seed;
	uint32_t current_target = 0;
	while (current_seed < seeds_max) {
		for (uint8_t i = 0; i < seed_offsets_len; i++) {
			if (!current_target)
				//don't return here to avoid complicated flow control, out of bound results will be filtered later
				current_seed = seeds_base++;
			JavaRandom gen(current_seed + seed_offsets[i]);
			if (gen.nextInt<10>()) current_target = 0;
			else if (++current_target == seed_offsets_len) {
				current_target = 0;
				good_seed = current_seed;
			}
		}
		if (good_seed != bad_seed && good_seed < seeds_max) {
			passed_seeds[atomicAdd(&passed_buffer_i, 1) & PASSED_BUFF_MASK] = good_seed;
			good_seed = bad_seed;
		}
	}
}

void launch_test_seeds(int blocks, int thread, hipStream_t s, uint64_t start, uint64_t end, uint64_t* passed_seeds) {
	test_seeds<<<blocks, thread, 0, s>>>(start, end, passed_seeds);
	hipGetLastError() && assertcu;
}

}
